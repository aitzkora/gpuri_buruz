
#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>
#define N 713
#define BLOCK_SIZE 256

__global__ void kernelXOR(int * a) 
{ 
  int  currentIndex = threadIdx.x + blockDim.x * blockIdx.x; 
  if (currentIndex < N) 
    a[currentIndex] ^= 0xFF;
}

int main()
{
  std::vector<int> tab(N, 1);
  int * tabGPU;
  hipMalloc(&tabGPU, N * sizeof(int));
  hipMemcpy(tabGPU, tab.data(), N * sizeof(int), hipMemcpyHostToDevice);
  kernelXOR<<<(N-1)/BLOCK_SIZE + 1, BLOCK_SIZE>>>(tabGPU);
  hipMemcpy(tab.data(), tabGPU, N * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(tabGPU);
  for(int i=0; i < N; ++i) 
    assert (tab[i] == 0xFE) ;
  return 0;
}
