
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 256

__global__ void increment(int * u, const int n, int val) {
  int i = threadIdx.x; u[i] += val;
}

int main()
{
  int tab[N];
  int * tab_d;
  int sizeCpy = N * sizeof(int);
  for(int i = 0; i < N; ++i)   tab[i] = 1;
  hipMalloc(&tab_d, sizeCpy);
  hipMemcpy(tab_d, tab, sizeCpy, hipMemcpyHostToDevice );
  increment<<<1,N>>>(tab_d, N, 3);
  hipMemcpy(tab, tab_d, sizeCpy, hipMemcpyDeviceToHost );
  hipFree(tab_d);
  for (int i = 0; i < N; ++i) { if (i[tab] != 4) abort();}
  return 0;
}
